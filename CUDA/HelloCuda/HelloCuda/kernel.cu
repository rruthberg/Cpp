
#include "hip/hip_runtime.h"


#include <stdio.h>




void regVecAdd(int *a, int *b, int *c, int n) {
	int i;
	for (i = 0; i < n; ++i)
		c[i] = a[i] + b[i];
}

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
	{
		c[i] = a[i] + b[i];
	}
}

int main() {
	hipError_t err = hipSuccess;
	int numelements = 1024;
	size_t SIZE = numelements * sizeof(int);
	printf("[Vector addition of %d elements]\n", numelements);
	
	//Define pointers
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	//Allocate memory on the host
	a = (int *)malloc(SIZE);
	b = (int *)malloc(SIZE);
	c = (int *)malloc(SIZE); 

	//Allocate memory on the device
	err = hipMalloc((void**)&d_a, SIZE);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void**)&d_b, SIZE);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void**)&d_c, SIZE);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	for (int i = 0; i < numelements; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	//Copy data from host to device
	err = hipMemcpy(d_a, a, SIZE, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to A - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_b, b, SIZE, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to B - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_c, c, SIZE, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to C - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}



	vectorAdd<<<1, numelements>>>(d_a, d_b, d_c, numelements); //call specifies blocks and threads by <<< BLOCKS, THREADS >>> so SIZE is the number of threads

	//copy data from device to host
	err = hipMemcpy(a, d_a, SIZE, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to A - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(b, d_b, SIZE, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to B - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(c, d_c, SIZE, hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to C - (error code %s)!", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	//Free memory on host and device
	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}